#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <memory>
#include "hip/hip_runtime.h"
__global__ void add_two_vectors(int* v1, int* v2, int* result){
  int idx = threadIdx.x;
  result[idx] = v1[idx] + v2[idx];
  //printf("%i, ",result[idx]);
}
int main(int argc, char **argv) {
    int* v1_host = (int*)malloc(64*sizeof(int));
    int* v2_host = (int*)malloc(64*sizeof(int));
    int* result_host = (int*)malloc(64*sizeof(int));
    //memset(v1_host,64,1);
    //memset(v2_host,64,2);
    for(int i=0;i<64;i++){
      v1_host[i] = 1;
      v2_host[i] = 2;
    }
    int* v1_dev;
    int* v2_dev;
    int* result_dev;
    hipMalloc(&v1_dev, 64*sizeof(int));
    hipMalloc(&v2_dev, 64*sizeof(int));
    hipMalloc(&result_dev, 64*sizeof(int));
    //copy memory 
    hipMemcpy(v1_dev,v1_host,64*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(v2_dev,v2_host,64*sizeof(int),hipMemcpyHostToDevice);
    //launch kernel 
    dim3 grid(1,1,1);
    dim3 block(64,1,1);
    add_two_vectors<<<grid, block>>>(v1_dev, v2_dev, result_dev);
    //sync
    hipDeviceSynchronize();
    //copy memory from device to host 
    hipMemcpy(result_host, result_dev, 64*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<64;i++){
        printf("%i, ",result_host[i]);
    }
    free(v1_host);
    free(v2_host);
    free(result_host);
    hipFree(v1_dev);
    hipFree(v2_dev);
    hipFree(result_dev);
    std::shared_ptr<int> iptr = std::make_shared<int>(4);
    printf("%i",*iptr);
}
