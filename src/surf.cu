#include "hip/hip_runtime.h"
#include "surf_cuda/common.h"
#include "surf_cuda/cuda_util.cuh"
#include "surf_cuda/surf.h"
namespace surf_cuda{
#if 0
SURF::SURF(){
  //first level octave 
  octaves_[0] = Octave(4,rows_,cols_,{9,15,21,27});
  
}
#endif
template <typename T>
__global__ void compRowIntegral(unsigned char* mat_in, unsigned char* mat_out, size_t rows, size_t cols, size_t pitch_bytes){
  int row_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(row_idx<rows){
        T* row_addr_in = (T*)(mat_in + row_idx*pitch_bytes);
        T* row_addr_out = (T*)(mat_out + row_idx*pitch_bytes);
	//compute integral along the row 
        T integral_cache;
//#pragma unroll 4
	for(size_t c=0;c<cols; c++){
            if(c==0){
                //initial value
                integral_cache=row_addr_in[c];
                row_addr_out[c]=integral_cache;
            } else{
                integral_cache = row_addr_in[c] + integral_cache;
                row_addr_out[c] = integral_cache;
            }
        }
    }
}
template <typename T>
__global__ void compColIntegral(unsigned char* mat_in, unsigned char* mat_out, size_t rows, size_t cols, size_t pitch_bytes){
  int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(col_idx<cols){
    T integral_cache;
//#pragma unroll 4
    for(int r=0; r<rows; r++){
      //compute row address
      T* row_addr_in = (T*)(mat_in + r*pitch_bytes);
      T* row_addr_out = (T*)(mat_out + r*pitch_bytes);
      if(r==0){
	integral_cache = row_addr_in[col_idx];
	row_addr_out[col_idx] = integral_cache;
      } else{
	integral_cache = row_addr_in[col_idx]+integral_cache;
	row_addr_out[col_idx] = integral_cache;
      }
    }
  }
}

void SURF::compIntegralImage(const CudaMat& img_in, const CudaMat& img_integral){
  printf("[CUDA] Computing Integral Image\n");
  //first compute integral along rows
  size_t block_dim_x_row = 128;
  size_t block_dim_x_col = 128;
  dim3 block_row(block_dim_x_row,1,1);
  dim3 grid_row(img_in.cols()/block_dim_x_row + 1,1,1);
  compRowIntegral<int> <<<block_row,grid_row>>>(img_in.data(), img_integral.data(),img_in.rows(), img_in.cols(), img_in.pitch_bytes());
  //sync
  hipDeviceSynchronize();
  //then compute integral along cols
  dim3 block_col(block_dim_x_col,1,1);
  dim3 grid_col(img_in.rows()/block_dim_x_col + 1,1,1);
  compColIntegral<int> <<<block_row, grid_row>>>(img_integral.data(), img_integral.data(),img_integral.rows(), img_integral.cols(), img_integral.pitch_bytes());
  hipDeviceSynchronize();
}
}