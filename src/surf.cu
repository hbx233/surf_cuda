#include "hip/hip_runtime.h"
#include "surf_cuda/common.h"
#include "surf_cuda/cuda_util.cuh"
#include "surf_cuda/surf.h"
namespace surf_cuda{
  
template <typename T>
__global__ void compRowIntegral(unsigned char* mat_in, unsigned char* mat_out, size_t rows, size_t cols, size_t pitch_bytes){
  int row_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(row_idx<rows){
        T* row_addr_in = (T*)(mat_in + row_idx*pitch_bytes);
        T* row_addr_out = (T*)(mat_out + row_idx*pitch_bytes);
	//compute integral along the row 
        T integral_cache;
//#pragma unroll 4
	for(size_t c=0;c<cols; c++){
            if(c==0){
                //initial value
                integral_cache=row_addr_in[c];
                row_addr_out[c]=integral_cache;
            } else{
                integral_cache = row_addr_in[c] + integral_cache;
                row_addr_out[c] = integral_cache;
            }
        }
    }
}
template <typename T>
__global__ void compColIntegral(unsigned char* mat_in, unsigned char* mat_out, size_t rows, size_t cols, size_t pitch_bytes){
  int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(col_idx<cols){
    T integral_cache;
//#pragma unroll 4
    for(int r=0; r<rows; r++){
      //compute row address
      T* row_addr_in = (T*)(mat_in + r*pitch_bytes);
      T* row_addr_out = (T*)(mat_out + r*pitch_bytes);
      if(r==0){
	integral_cache = row_addr_in[col_idx];
	row_addr_out[col_idx] = integral_cache;
      } else{
	integral_cache = row_addr_in[col_idx]+integral_cache;
	row_addr_out[col_idx] = integral_cache;
      }
    }
  }
}

void SURF::compIntegralImage(const CudaMat& img_in, const CudaMat& img_integral){
  printf("[CUDA] Computing Integral Image\n");
  //first compute integral along rows
  size_t block_dim_x_row = 128;
  size_t block_dim_x_col = 128;
  dim3 block_row(block_dim_x_row,1,1);
  dim3 grid_row(img_in.cols()/block_dim_x_row + 1,1,1);
  compRowIntegral<int> <<<block_row,grid_row>>>(img_in.data, img_integral.data,img_in.rows(), img_in.cols(), img_in.pitch_bytes());
  //sync
  hipDeviceSynchronize();
  //then compute integral along cols
  dim3 block_col(block_dim_x_col,1,1);
  dim3 grid_col(img_in.rows()/block_dim_x_col + 1,1,1);
  compColIntegral<int> <<<block_row, grid_row>>>(img_integral.data, img_integral.data,img_integral.rows(), img_integral.cols(), img_integral.pitch_bytes());
  hipDeviceSynchronize();
}

__global__ void kernel_DoH_Filter(unsigned char* integral_mat, size_t integral_pitch_bytes, int integral_rows, int integral_cols, unsigned char* response_mat, size_t response_pitch_bytes, int response_rows, int response_cols, int stride, DoHFilter doh_filter){
  //first check if the output response map with current stride parameter can fit in the memory of input response map
  //will not use the fast few columns for sub sampling 
  if(response_rows==integral_rows/stride || response_cols==integral_cols/stride){
    //Just one kernel per row's computation 
    int row_response_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(row_response_idx<response_rows){
      //response map's row pointer 
      float* row_response_addr = (float*)(response_mat + row_response_idx * response_pitch_bytes);
      int row_integral_idx = row_response_idx * stride;
      //loop through columns 
//#pragma unroll 4
      for(int c=0; c<response_cols; c++){
	row_response_addr[c] = doh_filter(integral_mat, integral_pitch_bytes, row_integral_idx, c*stride, integral_rows, integral_cols); 
      }
    }
  }
}

void SURF::compDoHBlobResponseMap(const CudaMat& img_integral, const CudaMat& img_doh_response, const DoHFilter& doh_filter ,const int& stride){
  //check CudaMat type 
  if(img_integral.type()==CV_32S && img_doh_response.type() == CV_32F){
    size_t block_dim_x = 128;
    dim3 block(block_dim_x,1,1);
    dim3 grid(img_doh_response.rows()/block_dim_x + 1,1,1);
    kernel_DoH_Filter<<<grid,block>>> (img_integral.data,img_integral.pitch_bytes(),img_integral.rows(),img_integral.cols(),img_doh_response.data,img_doh_response.pitch_bytes(),img_doh_response.rows(),img_doh_response.cols(),stride, doh_filter);
    CudaCheckError();
    hipDeviceSynchronize();
  } else{
    fprintf(stderr,"[CUDA] [DoH Response Map] The CudaMat type should be CV_32S for inpute, CV_32F for output");
  }
}




}