#include "surf_cuda/cuda_mat.h"

namespace surf_cuda{

void CudaMat::allocate()
{
  hipError_t err;
  bool leagal_type=true;
  switch(type_){
    //8bit unsigned char 
    case CV_8U:
      err = CudaSafeCall(hipMallocPitch((void**)&data,&pitch_bytes_,cols_*sizeof(unsigned char),rows_));
      depth_ = sizeof(unsigned char);
      elemSize_ = depth_;
      break;
    case CV_32S:
      err = CudaSafeCall(hipMallocPitch((void**)&data,&pitch_bytes_,cols_*sizeof(int),rows_));
      depth_ = sizeof(int);
      elemSize_ = depth_;
      break;
    case CV_32F:
      err = CudaSafeCall(hipMallocPitch((void**)&data,&pitch_bytes_,cols_*sizeof(float),rows_));
      depth_ = sizeof(float);
      elemSize_ = depth_;
      break;
    case CV_64F:
      err = CudaSafeCall(hipMallocPitch((void**)&data,&pitch_bytes_,cols_*sizeof(double),rows_));
      depth_ = sizeof(double);
      elemSize_ = depth_;
      break;
    default:
      //TODO: Error handling 
      fprintf(stderr,"Unsupported depth");
      leagal_type=false;
      exit(-1);
      break;
  }
  if(err==hipSuccess && leagal_type==true){
    internalAllocated_ = true;
  }
}

void CudaMat::allocateArray(){
  hipError_t err;
  bool leagal_type=true;
  hipChannelFormatDesc channelDesc;
  
  switch(type_){
    //8bit unsigned char 
    case CV_8U:
      channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
      err = CudaSafeCall(hipMallocArray(&cuda_array_, &channelDesc, cols_, rows_));
      channel_desc_ = channelDesc;
      break;
    case CV_32S:
      channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
      err = CudaSafeCall(hipMallocArray(&cuda_array_, &channelDesc, cols_, rows_));
      channel_desc_ = channelDesc;
      break;
    case CV_32F:
      channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
      err = CudaSafeCall(hipMallocArray(&cuda_array_, &channelDesc, cols_, rows_));
      channel_desc_ = channelDesc;
      break;
    case CV_64F:
      fprintf(stderr, "[CUDA] 64bit Float type is not supported for CUDA Texture Memory");
      leagal_type = false;
      exit(-1);
      break;
    default:
      //TODO: Error handling 
      fprintf(stderr,"Unsupported type");
      leagal_type=false;
      exit(-1);
      break;
  }
  if(err==hipSuccess && leagal_type==true){
    //set resourse descriptor
    memset(&res_desc_,0,sizeof(res_desc_));
    res_desc_.resType = hipResourceTypeArray;
    res_desc_.res.array.array = cuda_array_;
    internalAllocatedArray_ = true;
  }
  
}

void CudaMat::setTextureObjectInterface(hipTextureDesc tex_desc){
  tex_obj_ = 0;
  //store texture descriptor
  tex_desc_ = tex_desc;
  //create texture object 
  hipCreateTextureObject(&tex_obj_, &res_desc_, &tex_desc_, NULL);
}

void CudaMat::copyToArray(){
  //copy internal data in Global Memory to Texture Memory
  hipMemcpy2DToArray(cuda_array_, 0, 0, (void*)data, pitch_bytes_, cols_ * depth_, rows_, hipMemcpyDeviceToDevice);
}

void CudaMat::writeDevice(void* hostmem, size_t hostpitch_bytes, int width, int height)
{
  if(data==NULL){
    fprintf(stderr,"[CUDA] [Write Device], data not allocated\n");
    return;
  }
  if(width==cols_ && height==rows_){
    hipError_t err = CudaSafeCall(hipMemcpy2D((void*)data, pitch_bytes_, (void*)hostmem, hostpitch_bytes, width*elemSize_, height, hipMemcpyHostToDevice));
    if(err==hipSuccess){
      printf("[CUDA] Wrote %i bytes data to Device\n", width*height*elemSize_);
    } 
  } else{
    fprintf(stderr,"[CUDA] [Write Device] Dimension of Host Source Memory and Device Memory do not Match\n");
    fprintf(stderr,"[CUDA] Host   Memory width (in element): %i, height(in element): %i \n",width, height);
    fprintf(stderr,"[CUDA] Device Memory width (in element): %i, height(in element): %i \n",cols_, rows_);
  }
}


void CudaMat::writeDeviceToArray(void* hostmem, size_t hostpitch_bytes, int width, int height)
{
  if(data==NULL){
    fprintf(stderr,"[CUDA] [Write Device], data not allocated\n");
    return;
  }
  if(width==cols_ && height==rows_){
    hipError_t err = CudaSafeCall(hipMemcpy2DToArray(cuda_array_, 0, 0, hostmem, hostpitch_bytes, width * elemSize_, height, hipMemcpyHostToDevice));
    if(err==hipSuccess){
      printf("[CUDA] Wrote %i bytes data to Device\n", width*height*elemSize_);
    } 
  } else{
    fprintf(stderr,"[CUDA] [Write Device] Dimension of Host Source Memory and Device Memory do not Match\n");
    fprintf(stderr,"[CUDA] Host   Memory width (in element): %i, height(in element): %i \n",width, height);
    fprintf(stderr,"[CUDA] Device Memory width (in element): %i, height(in element): %i \n",cols_, rows_);
  }
}

void CudaMat::readDevice(void* hostmem, size_t hostpitch_bytes, int width, int height){
  if(data==NULL){
    fprintf(stderr,"[CUDA] [Read Device], data not allocated\n");
    return;
  }
  if(width==cols_ && height==rows_){
    hipError_t err = CudaSafeCall(hipMemcpy2D((void*)hostmem,hostpitch_bytes, (void*)data, pitch_bytes_, width*elemSize_,height, hipMemcpyDeviceToHost));
    if(err == hipSuccess){
      printf("[CUDA] Read %i bytes data from Device\n", width*height*elemSize_);
    }
  } else{
    fprintf(stderr,"[CUDA] [Read Device] Dimension of Device Source Memory and Source Memory do not Match\n");
    fprintf(stderr,"[CUDA] Host   Memory width (in element): %i, height(in element): %i \n",width, height);
    fprintf(stderr,"[CUDA] Device Memory width (in element): %i, height(in element): %i \n",cols_, rows_);
    exit(-1);
  }
}


void CudaMat::readDeviceFromArray(void* hostmem, size_t hostpitch_bytes, int width, int height){
  if(data==NULL){
    fprintf(stderr,"[CUDA] [Read Device], data not allocated\n");
    return;
  }
  if(width==cols_ && height==rows_){
    hipError_t err = CudaSafeCall(hipMemcpy2DFromArray((void*)hostmem, hostpitch_bytes, cuda_array_, 0, 0, width*elemSize_,height, hipMemcpyDeviceToHost));
    if(err == hipSuccess){
      printf("[CUDA] Read %i bytes data from Device\n", width*height*elemSize_);
    }
  } else{
    fprintf(stderr,"[CUDA] [Read Device] Dimension of Device Source Memory and Source Memory do not Match\n");
    fprintf(stderr,"[CUDA] Host   Memory width (in element): %i, height(in element): %i \n",width, height);
    fprintf(stderr,"[CUDA] Device Memory width (in element): %i, height(in element): %i \n",cols_, rows_);
    exit(-1);
  }
}

void CudaMat::copyFromMat(const Mat& mat)
{
  if(mat.type()==type_){
    writeDevice((void*)mat.data, mat.step[0], mat.cols, mat.rows);
  } else{
    fprintf(stderr, "[CUDA] [Write Device] Mat type not compatible");
    exit(-1);
  }
}
void CudaMat::copyToMat(Mat& mat)
{
  if(mat.type()==type_){
    mat = Mat(rows_, cols_, type_);
    readDevice((void*)mat.data, mat.step[0], mat.cols, mat.rows);
  } else{
    fprintf(stderr, "[CUDA] [Read device] Mat type not compatible");
  }
}



__host__ __device__ const int CudaMat::rows() const{
  return rows_;
}
__host__ __device__ const int CudaMat::cols() const{
  return cols_;
}
__host__ __device__ const size_t CudaMat::pitch_bytes() const{
  return pitch_bytes_;
}
__host__ __device__ const int CudaMat::depth() const{
  return depth_;
}
__host__ __device__ const int CudaMat::type() const{
  return type_;
}
__host__ __device__ const int CudaMat::elemSize() const{
  return elemSize_;
}
__host__ __device__ const hipTextureObject_t CudaMat::texture_object() const{
  return tex_obj_;
}
__host__ __device__ const hipChannelFormatDesc CudaMat::channel_desc() const{
  return channel_desc_;
}
__host__ __device__ const hipResourceDesc CudaMat::resource_desc() const{
  return res_desc_;
}
__host__ __device__ const hipTextureDesc CudaMat::texture_desc() const{
  return tex_desc_;
}
}